#include "hip/hip_runtime.h"
#include "temperature_controller_op/update_temperature_op.h"
#include "base/rocm.h"

namespace op
{
    #define THREADS_PER_BLOCK 256

	__global__
		void ComputeTemperature(const rbmd::Id& num_atoms,
			                    const rbmd::Real& mvv2e,
			                    const rbmd::Real* mass,
			                    const rbmd::Real* vx,
			                    const rbmd::Real* vy,
			                    const rbmd::Real* vz,
			                    rbmd::Real& temp_sum)
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		
		if (tid < num_atoms)
		{
			temp_sum += mvv2e * mass[tid] * (vx[tid] * vx[tid] + vy[tid] * vy[tid] + vz[tid] * vz[tid]);
		}
	}

	__global__
		void UpdataVelocityRescale(const rbmd::Id& num_atoms,
			                       const rbmd::Real& coeff_rescale,
			                       rbmd::Real* vx,
			                       rbmd::Real* vy,
			                       rbmd::Real* vz)
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid < num_atoms)
		{
			vx[tid] = vx[tid] * coeff_rescale;
			vy[tid] = vy[tid] * coeff_rescale;
			vz[tid] = vz[tid] * coeff_rescale;
		}
	}

	__global__
		void UpdataVelocityNoseHoover(const rbmd::Id& num_atoms,
			                          const rbmd::Real& dt,
			                          const rbmd::Real& fmt2v,
			                          const rbmd::Real& nosehooverxi,
			                          const rbmd::Real* mass
			                          const rbmd::Real* fx,
			                          const rbmd::Real* fy,
			                          const rbmd::Real* fz,
			                          rbmd::Real* vx,
			                          rbmd::Real* vy,
			                          rbmd::Real* vz)
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid < num_atoms)
		{
			vx[tid] += 0.5 * _dt * (fx / mass[tid] - nosehooverxi * vx[tid]) * fmt2v;
			vy[tid] += 0.5 * _dt * (fy / mass[tid] - nosehooverxi * vy[tid]) * fmt2v;
			vz[tid] += 0.5 * _dt * (fz / mass[tid] - nosehooverxi * vz[tid]) * fmt2v;
		}
	}

	__global__
		void UpdataVelocityRescale(const rbmd::Id& num_atoms,
			                       const rbmd::Real& coeff_Berendsen,
			                       rbmd::Real* vx,
			                       rbmd::Real* vy,
			                       rbmd::Real* vz)
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid < num_atoms)
		{
			vx[tid] = vx[tid] * coeff_Berendsen;
			vy[tid] = vy[tid] * coeff_Berendsen;
			vz[tid] = vz[tid] * coeff_Berendsen;
		}
	}

	struct ComputeTemperatureOp<device::DEVICE_GPU>
	{
		void operator()(const rbmd::Id& num_atoms,
			            const rbmd::Real& mvv2e,
			            const rbmd::Real* mass,
			            const rbmd::Real* vx,
			            const rbmd::Real* vy,
			            const rbmd::Real* vz,
			            rbmd::Real& temp_sum)
		{
			int block_per_grid = (nAtoms + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

			hipLaunchKernelGGL(HIP_KERNEL_NAME(ComputeTemperature), dim3(block_per_grid), dim3(THREADS_PER_BLOCK), 0, 0,
				num_atoms, mvv2e, mass, vx, vy, vz, temp_sum);

			hipErrorCheck(hipGetLastError());
			hipErrorCheck(hipDeviceSynchronize());
		}
	};


	struct UpdataVelocityRescaleOp<device::DEVICE_GPU>
	{
		void operator()(const rbmd::Id& num_atoms,
			            const rbmd::Real& coeff_rescale,
			            rbmd::Real* vx,
			            rbmd::Real* vy,
			            rbmd::Real* vz)
		{
			int block_per_grid = (nAtoms + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

			hipLaunchKernelGGL(HIP_KERNEL_NAME(UpdataVelocityRescale), dim3(block_per_grid), dim3(THREADS_PER_BLOCK), 0, 0,
				num_atoms, coeff_rescale, vx, vy, vz);

			hipErrorCheck(hipGetLastError());
			hipErrorCheck(hipDeviceSynchronize());
		}
	};


	struct UpdataVelocityNoseHooverOp<device::DEVICE_GPU>
	{
		void operator()(const rbmd::Id& num_atoms,
			            const rbmd::Real& dt,
			            const rbmd::Real& fmt2v,
			            const rbmd::Real& nosehooverxi,
			            const rbmd::Real* mass,
			            const rbmd::Real* fx,
			            const rbmd::Real* fy,
			            const rbmd::Real* fz,
			            rbmd::Real* vx,
			            rbmd::Real* vy,
			            rbmd::Real* vz) 
		{
			int block_per_grid = (nAtoms + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

			hipLaunchKernelGGL(HIP_KERNEL_NAME(UpdataVelocityNoseHoover), dim3(block_per_grid), dim3(THREADS_PER_BLOCK), 0, 0,
				num_atoms, dt, fmt2v, nosehooverxi, mass, fx, fy, fz, vx, vy, vz);

			hipErrorCheck(hipGetLastError());
			hipErrorCheck(hipDeviceSynchronize());

		}
	};

	struct UpdataVelocityBerendsenOp<device::DEVICE_GPU>
	{
		void operator()(const rbmd::Id& num_atoms,
			            const rbmd::Real& coeff_Berendsen,
			            rbmd::Real* vx,
			            rbmd::Real* vy,
			            rbmd::Real* vz)
		{
			int block_per_grid = (nAtoms + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

			hipLaunchKernelGGL(HIP_KERNEL_NAME(UpdataVelocityBerendsen), dim3(block_per_grid), dim3(THREADS_PER_BLOCK), 0, 0,
				num_atoms, coeff_Berendsen, vx, vy, vz);

			hipErrorCheck(hipGetLastError());
			hipErrorCheck(hipDeviceSynchronize());
		}
	};

	template struct ComputeTemperatureOp<device::DEVICE_GPU>;
	template struct UpdataVelocityRescaleOp<device::DEVICE_GPU>;
	template struct UpdataVelocityNoseHooverOp<device::DEVICE_GPU>
	template struct UpdataVelocityBerendsenOp<device::DEVICE_GPU>



}


#include "hip/hip_runtime.h"
#include "update_velocity_op.h"
#include "rbmd_define.h"

namespace op
{
    #define THREADS_PER_BLOCK 256

	__global__
		void UpdateVelocity(const rbmd::Id num_atoms,
			                const rbmd::Real dt,
			                const rbmd::Real fmt2v,
					        const rbmd::Id* atoms_type,
			                const rbmd::Real* fx,
			                const rbmd::Real* fy,
			                const rbmd::Real* fz,
			                const rbmd::Real* mass,
			                rbmd::Real* vx,
			                rbmd::Real* vy,
			                rbmd::Real* vz,
			                rbmd::Real* px,
			                rbmd::Real* py,
			                rbmd::Real* pz )
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid < num_atoms)
		{
			rbmd::Id typei = atoms_type[tid]-1; 
			rbmd::Real massi = 1.0;

			//printf("--------test---massi:%f---\n",massi);

			 //�����ٶ�
			vx[tid] += 0.5 * fx[tid] / massi * dt * fmt2v;
			vy[tid] += 0.5 * fy[tid] / massi * dt * fmt2v;
			vz[tid] += 0.5 * fz[tid] / massi * dt * fmt2v;

			if (tid == 0) {
				printf("--------test-dt:%f,--fmt2v:%f,---fx[tid]:%f, _vx:%f,%f,_vy:%f,_vz:%f,---\n", dt, fmt2v, fx[tid],vx[tid], vy[tid], vz[tid]);
			}
		}

	}


	void UpdateVelocityOp<device::DEVICE_GPU>::operator()(const rbmd::Id num_atoms,
			                                              const rbmd::Real dt,
			                                              const rbmd::Real fmt2v,
														  const rbmd::Id* atoms_type,
			                                              const rbmd::Real* mass,
			                                              const rbmd::Real* fx,
			                                              const rbmd::Real* fy,
			                                              const rbmd::Real* fz,
			                                              rbmd::Real* vx,
			                                              rbmd::Real* vy,
			                                              rbmd::Real* vz,
														  rbmd::Real* px,
															rbmd::Real* py,
															rbmd::Real* pz)
	{
		//printf("--------test---mass:%f---\n",mass[0]);
		unsigned int blocks_per_grid = (num_atoms + BLOCK_SIZE - 1) / BLOCK_SIZE;
		CHECK_KERNEL(UpdateVelocity <<<blocks_per_grid, BLOCK_SIZE, 0, 0 >>> (num_atoms, dt, fmt2v, atoms_type,mass, fx, fy, fz, vx, vy, vz, px, py, pz));

	}
}


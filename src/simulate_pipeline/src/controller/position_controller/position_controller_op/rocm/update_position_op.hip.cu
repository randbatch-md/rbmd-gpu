#include "hip/hip_runtime.h"
#include "update_position_op.h"
#include "rbmd_define.h"

namespace op
{
#define THREADS_PER_BLOCK 256

	__device__
		void UpdateFlagOverRangePoint(const rbmd::Real& min_x_tid,
			                          const rbmd::Real& min_y_tid,
			                          const rbmd::Real& min_z_tid,
			                          const rbmd::Real& max_x_tid,
			                          const rbmd::Real& max_y_tid,
			                          const rbmd::Real& max_z_tid,
                                      rbmd::Real& px_tid,
			                          rbmd::Real& py_tid,
			                          rbmd::Real& pz_tid,
			                          rbmd::Id& flag_px_tid,
			                          rbmd::Id& flag_py_tid,
			                          rbmd::Id& flag_pz_tid)
	{

		flag_px_tid += (px_tid > max_x_tid)-(px_tid < min_x_tid);
		px_tid += (px_tid < min_x_tid) * (max_x_tid - min_x_tid) - (px_tid > max_x_tid) * (max_x_tid - min_x_tid);

		flag_py_tid += (py_tid > max_y_tid) - (py_tid < min_y_tid);
		py_tid += (py_tid < min_y_tid) * (max_y_tid - min_y_tid) - (py_tid > max_y_tid) * (max_y_tid - min_y_tid);

		flag_pz_tid += (pz_tid > max_z_tid) - (pz_tid < min_z_tid);
		pz_tid += (pz_tid < min_z_tid) * (max_z_tid - min_z_tid) - (pz_tid > max_z_tid) * (max_z_tid - min_z_tid);
		/*
		if (px_tid < min_x_tid)
		{
			px_tid += max_x_tid - min_x_tid;
			flag_px_tid -= 1;
		}
		else if (px_tid > max_x_tid)
		{
			px_tid -= max_x_tid - min_x_tid;
			flag_px_tid += 1;
		}
		
		if (py_tid < min_y_tid)
		{
			py_tid += max_y_tid - min_y_tid;
			flag_py_tid -= 1;
		}
		else if (py_tid > max_y_tid)
		{
			py_tid -= max_y_tid - min_y_tid;
			flag_py_tid += 1;
		}

		if (pz_tid < min_z_tid)
		{
			pz_tid += max_z_tid - min_z_tid;
			flag_pz_tid -= 1;
		}
		else if (pz_tid > max_z_tid)
		{
			pz_tid -= max_z_tid - min_z_tid;
			flag_pz_tid += 1;
		}
		*/
	}

	__device__
		void UpdateOverRangePoint(const rbmd::Real& min_x_tid,
		                          const rbmd::Real& min_y_tid,
		                          const rbmd::Real& min_z_tid,
		                          const rbmd::Real& max_x_tid,
		                          const rbmd::Real& max_y_tid,
		                          const rbmd::Real& max_z_tid,
		                          rbmd::Real& px_tid,
		                          rbmd::Real& py_tid,
		                          rbmd::Real& pz_tid)
	{
		px_tid += (px_tid < min_x_tid) * (max_x_tid - min_x_tid) - (px_tid > max_x_tid) * (max_x_tid - min_x_tid);

		py_tid += (py_tid < min_y_tid) * (max_y_tid - min_y_tid) - (py_tid > max_y_tid) * (max_y_tid - min_y_tid);

		pz_tid += (pz_tid < min_z_tid) * (max_z_tid - min_z_tid) - (pz_tid > max_z_tid) * (max_z_tid - min_z_tid);
		
		/*
		if (px_tid < min_x_tid)
		{
			px_tid += max_x_tid - min_x_tid;
		}
		else if (px_tid > max_x_tid)
		{
			px_tid -= max_x_tid - min_x_tid;
		}

		if (py_tid < min_y_tid)
		{
			py_tid += max_y_tid - min_y_tid;
		}
		else if (py_tid > max_y_tid)
		{
			py_tid -= max_y_tid - min_y_tid;
		}

		if (pz_tid < min_z_tid)
		{
			pz_tid += max_z_tid - min_z_tid;
		}
		else if (pz_tid > max_z_tid)
		{
			pz_tid -= max_z_tid - min_z_tid;
		}
		*/
	}

	__global__
		void UpdatePositionFlag(const rbmd::Id& num_atoms,
			                    const rbmd::Real& dt,
			                    const rbmd::Real& min_x,
			                    const rbmd::Real& min_y,
			                    const rbmd::Real& min_z,
			                    const rbmd::Real& max_x,
			                    const rbmd::Real& max_y,
			                    const rbmd::Real& max_z,
			                    const rbmd::Real* vx,
			                    const rbmd::Real* vy,
			                    const rbmd::Real* vz,
			                    rbmd::Real* px,
			                    rbmd::Real* py,
			                    rbmd::Real* pz,
			                    rbmd::Id* flag_px,
			                    rbmd::Id* flag_py,
			                    rbmd::Id* flag_pz)
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid < num_atoms)
		{
			px[tid] += vx[tid] * dt;
			py[tid] += vy[tid] * dt;
			pz[tid] += vz[tid] * dt;

			UpdateFlagOverRangePoint(min_x,
				                     min_y,
				                     min_z,
				                     max_x,
				                     max_y,
				                     max_z,
				                     px[tid],
				                     py[tid],
				                     pz[tid],
				                     flag_px[tid],
				                     flag_py[tid],
				                     flag_pz[tid]);
		}
	}

	__global__
		void UpdatePosition(const rbmd::Id& num_atoms,
			                const rbmd::Real& dt,
			                const rbmd::Real& min_x,
			                const rbmd::Real& min_y,
			                const rbmd::Real& min_z,
			                const rbmd::Real& max_x,
			                const rbmd::Real& max_y,
			                const rbmd::Real& max_z,
			                const rbmd::Real* vx,
			                const rbmd::Real* vy,
			                const rbmd::Real* vz,
			                rbmd::Real* px,
			                rbmd::Real* py,
			                rbmd::Real* pz)
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid < num_atoms)
		{
			px[tid] += vx[tid] * dt;
			py[tid] += vy[tid] * dt;
			pz[tid] += vz[tid] * dt;

			UpdateOverRangePoint(min_x,
				                 min_y,
				                 min_z,
				                 max_x,
				                 max_y,
				                 max_z,
				                 px[tid],
				                 py[tid],
				                 pz[tid]);
		}
	}


	void UpdatePositionFlagOp<device::DEVICE_GPU>::operator()(const rbmd::Id& num_atoms,
			            const rbmd::Real& dt,
			            const rbmd::Real& min_x,
			            const rbmd::Real& min_y,
			            const rbmd::Real& min_z,
			            const rbmd::Real& max_x,
			            const rbmd::Real& max_y,
			            const rbmd::Real& max_z,
			            const rbmd::Real* vx,
			            const rbmd::Real* vy,
			            const rbmd::Real* vz,
			            rbmd::Real* px,
			            rbmd::Real* py,
			            rbmd::Real* pz,
			            rbmd::Id* flag_px,
			            rbmd::Id* flag_py,
			            rbmd::Id* flag_pz)
	{
		unsigned int blocks_per_grid = (num_atoms + BLOCK_SIZE - 1) / BLOCK_SIZE;
		CHECK_KERNEL(UpdatePositionFlag <<<blocks_per_grid, BLOCK_SIZE, 0, 0 >>> (num_atoms, dt, min_x, min_y, min_z, max_x, max_y, max_z, vx, vy, vz, px, py, pz, flag_px, flag_py, flag_pz));
	}
	

	void UpdatePositionOp<device::DEVICE_GPU>::operator()(const rbmd::Id& num_atoms,
			            const rbmd::Real& dt,
			            const rbmd::Real& min_x,
			            const rbmd::Real& min_y,
			            const rbmd::Real& min_z,
			            const rbmd::Real& max_x,
			            const rbmd::Real& max_y,
			            const rbmd::Real& max_z,
			            const rbmd::Real* vx,
			            const rbmd::Real* vy,
			            const rbmd::Real* vz,
			            rbmd::Real* px,
			            rbmd::Real* py,
			            rbmd::Real* pz)
	{
		unsigned int blocks_per_grid = (num_atoms + BLOCK_SIZE - 1) / BLOCK_SIZE;
		CHECK_KERNEL(UpdatePosition <<<blocks_per_grid, BLOCK_SIZE, 0, 0 >>> (num_atoms, dt, min_x, min_y, min_z, max_x, max_y, max_z, vx, vy, vz, px, py, pz));
	}
	
}
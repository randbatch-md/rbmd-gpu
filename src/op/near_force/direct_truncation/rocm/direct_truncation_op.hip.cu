#include "hip/hip_runtime.h"
#include "near_force/direct_truncation/direct_truncation_op.h"
#include "base/rocm.h"

namespace op
{

#define THREADS_PER_BLOCK 256

__device__
void ComputeCellId(
	const rbmd::Real3& position,
	rbmd::Id3& cellids,
	const rbmd::Real3& left,
	const rbmd::Real3& right,
	const rbmd::Id3& dim)
{
	__shared__ rbmd::Real3 dxdydz = (right - left) / dim; //should be shared memory
	cellids.data[0] = (position.data[0] - left.data[0]) / dxdydz.data[0];
	cellids.data[1] = (position.data[1] - left.data[1]) / dxdydz.data[1];
	cellids.data[2] = (position.data[2] - left.data[2]) / dxdydz.data[2];
}

template <typename FPTYPE>
__device__ __inline__
void UpdateVelocity()
{

}

template<typename FPTYPE>
__global__ 
void ComputeForce(
	const int& nAtoms,
	const FPTYPE* dt,
	const FPTYPE* fmt2v,
	const FPTYPE* mass,
	const rbmd::Real3& left,
	const rbmd::Real3& right,
	const rbmd::Id3& dim,
	rbmd::Id3* cellid,
	const Locator* locator,
	rbmd::Real3* position,
	rbmd::Real3* v,
	rbmd::Real3* force)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid > nAtoms)
	{
		return;
	}
	//cell id list
	ComputeCellId(position[tid], cellid[tid], left, right, dim);

	if (100 == tid)
	{
		printf("cell id: %d,%d,%d", cellid[tid].data[0], cellid[tid].data[1], cellid[tid].data[2]);
		printf("dt: %f\n", *dt);
		printf("fmt2v: %f\n", *fmt2v);
		printf("mass: %f\n", mass[0]);
		printf("v: %f\n", v[0].data[0]);
		printf("force: %f\n", force[0].data[0]);
	}
}


template<typename FPTYPE>
struct direct_truncation_op<FPTYPE, device::DEVICE_GPU>
{
	void operator()(
		const int& nSteps,
		const int& nAtoms,
		const rbmd::Real3& left, //shared memory
		const rbmd::Real3& right,
		const rbmd::Id3& dim,
		rbmd::Id3* cellid,
		const FPTYPE* dt,
		const FPTYPE* fmt2v,
		const FPTYPE* mass,
		const Locator* locator,
		rbmd::Real3* position,
		rbmd::Real3* v,
		rbmd::Real3* force)
	{
		int block = (nAtoms + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

		printf("nSteps: %d\n", nSteps);
		printf("nAtoms: %d\n", nAtoms);



		hipLaunchKernelGGL(HIP_KERNEL_NAME(ComputeForce<FPTYPE>), dim3(block), dim3(THREADS_PER_BLOCK), 0, 0,
			nAtoms, dt, fmt2v, mass, left, right, dim, cellid, locator, position, v, force);


		hipErrorCheck(hipGetLastError());
		hipErrorCheck(hipDeviceSynchronize());
	}
};

template struct direct_truncation_op<float, device::DEVICE_GPU>;
template struct direct_truncation_op<double, device::DEVICE_GPU>;
}


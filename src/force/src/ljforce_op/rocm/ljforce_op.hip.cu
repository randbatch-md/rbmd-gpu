#include "hip/hip_runtime.h"
#include "ljforce_op.h"
#include "model/box.h"
#include <hip/hip_runtime.h>
#include "../common/rbmd_define.h"

namespace op
{

	//force kernel 
    //LJForce
    __device__ void lj126(
		    rbmd::Real cut_off,
		    rbmd::Real px12,
		    rbmd::Real py12,
		    rbmd::Real pz12,
		    rbmd::Real eps_ij,
		    rbmd::Real sigma_ij,
			rbmd::Real& f_ij,
			rbmd::Real& e_ij)
	{
		const rbmd::Real  small_value = 0.0001;
		const rbmd::Real dis_2 = px12 * px12 + py12 * py12 + pz12 * pz12;
		const rbmd::Real cut_off_2 = cut_off * cut_off;

		if (dis_2 < cut_off_2 && dis_2 > small_value)
		{
			rbmd::Real sigmaij_6 = sigma_ij * sigma_ij * sigma_ij * sigma_ij * sigma_ij * sigma_ij;
			rbmd::Real dis_6 = dis_2 * dis_2 * dis_2;
			rbmd::Real sigmaij_dis_6 = sigmaij_6 / dis_6;


			 f_ij = -24 * eps_ij * ((2 * sigmaij_dis_6 - 1) * sigmaij_dis_6) / dis_2 ;
			 e_ij = 0.5 * (4 * eps_ij * (sigmaij_6 / dis_6 - 1) * (sigmaij_6 / dis_6));
		}
	}

	__device__ void lj126_rs(
		rbmd::Real rs,
		rbmd::Real px12,
		rbmd::Real py12,
		rbmd::Real pz12,
		rbmd::Real eps_ij,
		rbmd::Real sigma_ij,
		rbmd::Real& fs_ij)
	{
		const rbmd::Real  small_value = 0.0001;
		const rbmd::Real dis_2 = px12 * px12 + py12 * py12 + pz12 * pz12;
		const rbmd::Real rs_2 = rs * rs;

		if (dis_2 < rs_2 && dis_2 > small_value)
		{
			rbmd::Real sigmaij_6 = sigma_ij * sigma_ij * sigma_ij * sigma_ij * sigma_ij * sigma_ij;
			rbmd::Real dis_6 = dis_2 * dis_2 * dis_2;
			rbmd::Real sigmaij_dis_6 = sigmaij_6 / dis_6;

			fs_ij = -24 * eps_ij * ((2 * sigmaij_dis_6 - 1) * sigmaij_dis_6) / dis_2;
		}
	}

	__device__ void lj126_rcs(
		rbmd::Real rc,
		rbmd::Real rs,
		rbmd::Id pice_num,
		rbmd::Real px12,
		rbmd::Real py12,
		rbmd::Real pz12,
		rbmd::Real eps_ij,
		rbmd::Real sigma_ij,
		rbmd::Real& fcs_ij)
	{
		const rbmd::Real  small_value = 0.0001;
		const rbmd::Real dis_2 = px12 * px12 + py12 * py12 + pz12 * pz12;
		const rbmd::Real rc_2 = rc * rc;
		const rbmd::Real rs_2 = rs * rs;

		if (dis_2 < rc_2 && dis_2 > rs_2)
		{
			rbmd::Real sigmaij_6 = sigma_ij * sigma_ij * sigma_ij * sigma_ij * sigma_ij * sigma_ij;
			rbmd::Real dis_6 = dis_2 * dis_2 * dis_2;
			rbmd::Real sigmaij_dis_6 = sigmaij_6 / dis_6;

			fcs_ij = pice_num  * (-24 * eps_ij * ((2 * sigmaij_dis_6 - 1) * sigmaij_dis_6) / dis_2);
		}
	}

	__global__ void ComputeLJForce(
						Box* box,
			            const rbmd::Real cut_off,
			            const rbmd::Id num_atoms,
			            const rbmd::Id* atoms_type,
			            const rbmd::Id* molecular_type,
			            const rbmd::Real* sigma,
			            const rbmd::Real* eps,
			            const rbmd::Id* start_id,
		                const rbmd::Id* end_id,
		                const rbmd::Id* id_verletlist,
			            const rbmd::Real* px,
			            const rbmd::Real* py,
			            const rbmd::Real* pz,
			            rbmd::Real* force_x,
			            rbmd::Real* force_y,
			            rbmd::Real* force_z,
			            rbmd::Real* evdwl,
					    rbmd::Real* total_evdwl)
	{
		rbmd::Real sum_fx = 0;
		rbmd::Real sum_fy = 0;
	    rbmd::Real sum_fz = 0;
		rbmd::Real sum_eij = 0;

		unsigned int tid1 = blockIdx.x * blockDim.x + threadIdx.x;
		if (tid1 < num_atoms)
		{
			rbmd::Id typei = atoms_type[tid1]; 
			rbmd::Id molecular_id_i=  molecular_type[tid1];
			rbmd::Real eps_i = eps[typei];
			rbmd::Real sigma_i = sigma[typei];

			rbmd::Real x1 = px[tid1];
			rbmd::Real y1 = py[tid1];
			rbmd::Real z1 = pz[tid1];

			for (int j = start_id[tid1]; j < end_id[tid1]; ++j)
			{

				rbmd::Id tid2 = id_verletlist[j];
				rbmd::Id typej = atoms_type[tid2];
				rbmd::Id molecular_id_j = molecular_type[tid2];
				rbmd::Real eps_j = eps[typej];
				rbmd::Real sigma_j = sigma[typej];

				//mix
				rbmd::Real eps_ij = sqrt(eps_i * eps_j);
				rbmd::Real sigma_ij = (sigma_i + sigma_j) / 2;

				rbmd::Real x2 = px[tid2];
				rbmd::Real y2 = py[tid2];
				rbmd::Real z2 = pz[tid2];
				rbmd::Real px12 = x2 - x1;
				rbmd::Real py12 = y2 - y1;
				rbmd::Real pz12 = z2 - z1;
				//if (molecular_id_i == molecular_id_j)
					//continue; 

				MinImageDistance(box, px12, py12, pz12);

				rbmd::Real f_ij;
				rbmd::Real e_ij;

				lj126(cut_off, px12, py12, pz12, eps_ij, sigma_ij,f_ij,e_ij);
				sum_fx += f_ij * px12;
				sum_fy += f_ij * py12;
				sum_fz += f_ij * pz12;
				sum_eij += e_ij;
			}

			force_x[tid1] = sum_fx;
			force_y[tid1] = sum_fy;
			force_z[tid1] = sum_fz;

		    evdwl[tid1] += sum_eij;
			atomicAdd(total_evdwl, sum_eij);
			//printf("--------test---evdwl[tid1]:%f---\n",evdwl[tid1]);
		}
		
	}

	__global__ void ComputeLJRBLForce(
		Box* box,
		const rbmd::Real rs,
		const rbmd::Real rc,
		const rbmd::Id pice_num,
		const rbmd::Id num_atoms,
		const rbmd::Id* atoms_type,
		const rbmd::Id* molecular_type,
		const rbmd::Real* sigma,
		const rbmd::Real* eps,
		const rbmd::Id* start_id,
		const rbmd::Id* end_id,
		const rbmd::Id* id_verletlist,
		const rbmd::Id* id_verletlist_rcs,
		const rbmd::Real* px,
		const rbmd::Real* py,
		const rbmd::Real* pz,
		rbmd::Real* corr_force_x,
		rbmd::Real* corr_force_y,
		rbmd::Real* corr_force_z,
		rbmd::Real* corr_value_x,
		rbmd::Real* corr_value_y,
		rbmd::Real* corr_value_z)
	{
		rbmd::Real sum_fx = 0;
		rbmd::Real sum_fy = 0;
		rbmd::Real sum_fz = 0;
		rbmd::Real sum_eij = 0;

		rbmd::Real sum_fsx = 0;
		rbmd::Real sum_fsy = 0;
		rbmd::Real sum_fsz = 0;

		rbmd::Real sum_fcsx = 0;
		rbmd::Real sum_fcsy = 0;
		rbmd::Real sum_fcsz = 0;

		rbmd::Real  corr_fx_temp = 0;
		rbmd::Real  corr_fy_temp = 0;
		rbmd::Real  corr_fz_temp = 0;


		unsigned int tid1 = blockIdx.x * blockDim.x + threadIdx.x;
		if (tid1 < num_atoms)
		{
			rbmd::Id typei = atoms_type[tid1];
			rbmd::Id molecular_id_i = molecular_type[tid1];
			rbmd::Real eps_i = eps[typei];
			rbmd::Real sigma_i = sigma[typei];

			rbmd::Real x1 = px[tid1];
			rbmd::Real y1 = py[tid1];
			rbmd::Real z1 = pz[tid1];

			rbmd::Real fs_ij, fcs_ij;


			//rs 
			for (int j = start_id[tid1]; j < end_id[tid1]; ++j)
			{

				rbmd::Id tid2 = id_verletlist[j];
				rbmd::Id typej = atoms_type[tid2];
				rbmd::Id molecular_id_j = molecular_type[tid2];
				rbmd::Real eps_j = eps[typej];
				rbmd::Real sigma_j = sigma[typej];

				//mix
				rbmd::Real eps_ij = sqrt(eps_i * eps_j);
				rbmd::Real sigma_ij = (sigma_i + sigma_j) / 2;

				rbmd::Real x2 = px[tid2];
				rbmd::Real y2 = py[tid2];
				rbmd::Real z2 = pz[tid2];
				rbmd::Real px12 = x2 - x1;
				rbmd::Real py12 = y2 - y1;
				rbmd::Real pz12 = z2 - z1;
				//if (molecular_id_i == molecular_id_j)
					//continue; 

				MinImageDistance(box, px12, py12, pz12);

				lj126_rs(rs, px12, py12, pz12, eps_ij, sigma_ij, fs_ij);			// the force of rs 

				sum_fsx += fs_ij * px12;
				sum_fsy += fs_ij * py12;
				sum_fsz += fs_ij * pz12;
			}

			//rcs
			for (int j = id_verletlist_rcs[tid1* pice_num]; j < pice_num; ++j)
			{

				rbmd::Id tid2 = id_verletlist_rcs[j];
				rbmd::Id typej = atoms_type[tid2];
				rbmd::Id molecular_id_j = molecular_type[tid2];
				rbmd::Real eps_j = eps[typej];
				rbmd::Real sigma_j = sigma[typej];

				//mix
				rbmd::Real eps_ij = sqrt(eps_i * eps_j);
				rbmd::Real sigma_ij = (sigma_i + sigma_j) / 2;

				rbmd::Real x2 = px[tid2];
				rbmd::Real y2 = py[tid2];
				rbmd::Real z2 = pz[tid2];
				rbmd::Real px12 = x2 - x1;
				rbmd::Real py12 = y2 - y1;
				rbmd::Real pz12 = z2 - z1;
				//if (molecular_id_i == molecular_id_j)
					//continue; 

				MinImageDistance(box, px12, py12, pz12);

				lj126_rcs(rc, rs, pice_num,px12, py12, pz12, eps_ij, sigma_ij, fcs_ij); //the force  rs to rc

				sum_fcsx += fcs_ij * px12;
				sum_fcsy += fcs_ij * py12;
				sum_fcsz += fcs_ij * pz12;
			}

			//sum
			sum_fx = sum_fsx + sum_fcsx;
			sum_fy = sum_fsy + sum_fcsy;
			sum_fz = sum_fsz + sum_fcsz;
			
			corr_force_x[tid1] = sum_fx;
			corr_force_y[tid1] = sum_fy;
			corr_force_z[tid1] = sum_fz;

			corr_fx_temp += corr_force_x[tid1];
			corr_fy_temp += corr_force_y[tid1];
			corr_fz_temp += corr_force_z[tid1];

			corr_force_x[tid1] = corr_force_x[tid1] - corr_fx_temp;
			corr_force_y[tid1] = corr_force_y[tid1] - corr_fy_temp;
			corr_force_z[tid1] = corr_force_z[tid1] - corr_fz_temp;


			//atomicAdd(corr_value_x, corr_fx_temp);
			//atomicAdd(corr_value_y, corr_fy_temp);
			//atomicAdd(corr_value_z, corr_fz_temp);
			//printf("--------test---evdwl[tid1]:%f---\n",evdwl[tid1]);
		}

	}

	__device__
		rbmd::Real CoulForce(
			 rbmd::Real cut_off,
			 rbmd::Real alpha,
			 rbmd::Real charge_pi,
			 rbmd::Real charge_pj,
			 rbmd::Real px12,
			 rbmd::Real py12,
			 rbmd::Real pz12)
	{
		rbmd::Real MY_pi = 3.14159265358979323846; //pi
		rbmd::Real MY_pis = 1.77245385090551602729; // sqrt(pi)

		rbmd::Real CoulForce = 0;
		const rbmd::Real  small_value = 0.0001;
		const rbmd::Real dis_2 = px12 * px12 + py12 * py12 + pz12 * pz12;
		const rbmd::Real dis = sqrt(dis_2);
		const rbmd::Real cut_off_2 = cut_off * cut_off;

		if (dis_2 < cut_off_2 && dis_2 > small_value)
		{
			rbmd::Real erfcx = sqrt(alpha) * dis;
			rbmd::Real expx = -alpha * dis * dis;
			rbmd::Real Gnearvalue = (1.0 - erf(erfcx)) / (dis * dis) +
				2 * sqrt(alpha) * exp(expx) / (MY_pis * dis);

			CoulForce = -charge_pi * charge_pj * Gnearvalue / dis;


		}
		return CoulForce;
	}

	__device__ void EwaldForce(
		Box* box,
		const rbmd::Real alpha
		float3 M,
		float2 rhok_ri,
		const rbmd::Real* charge,
		const rbmd::Real* px,
		const rbmd::Real* py,
		const rbmd::Real* pz,
		rbmd::Real ewald_force_x,
		rbmd::Real ewald_force_y
		rbmd::Real ewald_force_z)
	{
		rbmd::Real ewald_force;
		rbmd::Real volume = box->_length[0] * box->_length[1] * box->_length[2];

		float3 k{ 0,0,0 };
		K[0] = 2 * M_PI * M[0] / box->_length[0];
		K[1] = 2 * M_PI * M[1] / box->_length[1];
		K[2] = 2 * M_PI * M[2] / box->_length[2];

		rbmd::Real range_K_2 = K[0] * K[0] + K[1] * K[1] + K[2] * K[2];
		rbmd::Real dot_product = K[0] * px + K[1] * py + K[2] * pz;

		rbmd::Real factor_a = -4 * M_PI * charge;
		rbmd::Real factor_b = exp(-range_K_2 / (4 * alpha));
		rbmd::Real factor_c = cos(dot_product) * rhok_ri[1];
		rbmd::Real factor_d = sin(dot_product) * rhok_ri[0];

		ewald_force = factor_a / (volume * range_K_2) * factor_b * (factor_c - factor_d);
		ewald_force_x = ewald_force * K[0];
		ewald_force_y = ewald_force * K[1];
		ewald_force_z = ewald_force * K[2];
	}


	__global__ void ComputeChargeStructureFactorComponent(
		const rbmd::Id num_atoms,
		float3 K,
		const rbmd::Real* px,
		const rbmd::Real* py,
		const rbmd::Real* pz,
		const rbmd::Real* charge,
		rbmd::Real* density_real,
		rbmd::Real* density_imag)
	{
		unsigned int tid1 = blockIdx.x * blockDim.x + threadIdx.x;
		if (tid1 < num_atoms)
		{
			rbmd::Real local_charge = charge[tid1];
			rbmd::Real dot_product = K.x * px[tid1] + K.y * py[tid1] + K.z * pz[tid1];

			density_real[tid1] = local_charge * cos(dot_product);
			density_imag[tid1] = local_charge * sin(dot_product);
		}
	}

	__global__ void ComputeEwaldForce(
		Box* box,
		const rbmd::Id num_atoms,
	    const rbmd::Id  Kmax,
		const rbmd::Real alpha,
	    const float2  whole_rhok,
		const rbmd::Real* charge,
		const rbmd::Real* px,
		const rbmd::Real* py,
		const rbmd::Real* pz,
		rbmd::Real* ewald_force_x,
		rbmd::Real* ewald_force_y,
		rbmd::Real* ewald_force_z)
	{
		unsigned int tid1 = blockIdx.x * blockDim.x + threadIdx.x;

		if (tid1 < num_atoms)
		{
			rbmd::Real p_x = px[tid1];
			rbmd::Real p_y = py[tid1];
			rbmd::Real p_z = pz[tid1];
			charge_i = charge[tid1];
			rbmd::Real force_x, force_y, force_z;

			for (rbmd::Id i = -Kmax; i <= Kmax; ++i)
			{
				for (rbmd::Id j = -Kmax; j <= Kmax; ++j)
				{
					for (rbmd::Id k = -Kmax; k <= Kmax; ++k)
					{
						if (!(i == 0 && j == 0 && k == 0)) 
						{
							float3 M = { (rbmd::Real)i, (rbmd::Real)j, (rbmd::Real)k };
							rbmd::Id indexEwald = (i + Kmax) * (2 * Kmax + 1) * 
												  (2 * Kmax + 1) + (j + Kmax) *
												  (2 * Kmax + 1) + (k + Kmax);
							 float2 rhok_i = whole_rhok[indexEwald];

							 EwaldForce(box, alpha,M, rhok_i, charge_i, p_x, p_y, p_z,
										force_x, force_y, force_z);
						}
					}
				}
			}
			ewald_force_x[tid1] = force_x;
			ewald_force_y[tid1] = force_y;
			ewald_force_z[tid1] = force_z;
	    }
	}


	//LJVirial
	//__global__ void ComputeJVirial(
	//		Box* box,
	//	    const rbmd::Real cut_off,
	//		const rbmd::Id& num_atoms,
	//		const rbmd::Id* atoms_type,
	//		const rbmd::Id* molecular_type,
	//		const rbmd::Real* eps,
	//		const rbmd::Real* sigma,
	//		const rbmd::Id* start_id,
	//		const rbmd::Id* end_id,
	//		const rbmd::Id* id_verletlist,
	//		const rbmd::Real* px,
	//		const rbmd::Real* py,
	//		const rbmd::Real* pz,
	//		rbmd::Real* virial_xx,
	//		rbmd::Real* virial_yy,
	//		rbmd::Real* virial_zz,
	//		rbmd::Real* virial_xy,
	//		rbmd::Real* virial_xz,
	//		rbmd::Real* virial_yz)
	//{

	//	rbmd::Real sum_virial_xx = 0;
	//	rbmd::Real sum_virial_yy= 0;
	//	rbmd::Real sum_virial_zz = 0;
	//	rbmd::Real sum_virial_xy = 0;
	//	rbmd::Real sum_virial_xz = 0;
	//	rbmd::Real sum_virial_yz = 0;

	//	int tid1 = threadIdx.x + blockIdx.x * blockDim.x;
	//	if (tid1 < num_atoms)
	//	{
	//		rbmd::Id typei = atoms_type[tid1];
	//		rbmd::Id molecular_id_i = molecular_type[tid1];
	//		rbmd::Real eps_i = eps[typei];
	//		rbmd::Real sigma_i = sigma[typei];

	//		rbmd::Real x1 = px[tid1];
	//		rbmd::Real y1 = py[tid1];
	//		rbmd::Real z1 = pz[tid1];


	//		for (int j = start_id[tid1]; j < end_id[tid1]; ++j)
	//		{
	//			rbmd::Id tid2 = id_verletlist[j];

	//			rbmd::Id typej = atoms_type[tid2];
	//			rbmd::Id molecular_id_j = molecular_type[tid2];
	//			rbmd::Real eps_j = eps[typej];
	//			rbmd::Real sigma_j = sigma[typej];

	//			//mix
	//			rbmd::Real eps_ij = sqrt(eps_i * eps_j);
	//			rbmd::Real sigma_ij = (sigma_i + sigma_j) / 2;

	//			rbmd::Real x2 = px[tid2];
	//			rbmd::Real y2 = py[tid2];
	//			rbmd::Real z2 = pz[tid2];
	//			rbmd::Real px12 = x2 - x1;
	//			rbmd::Real py12 = y2 - y1;
	//			rbmd::Real pz12 = z2 - z1;

	//			if (molecular_id_i == molecular_id_j)
	//				continue;

	//			//MinMirror(box, px12, py12, pz12);
	//			rbmd::Real Virial_f;
	//			Virial_f = LJVirial(cut_off, px12, py12, pz12, eps_ij, sigma_ij);

	//			rbmd::Real Virial_fx = Virial_f * px12;
	//			rbmd::Real Virial_fy = Virial_f * py12;
	//			rbmd::Real Virial_fz = Virial_f * pz12;

	//			sum_virial_xx +=  px12 * Virial_fx;
	//			sum_virial_yy +=  py12 * Virial_fy;
	//			sum_virial_zz +=  pz12 * Virial_fz;
	//			sum_virial_xy +=  px12 * Virial_fy;
	//			sum_virial_xz +=  px12 * Virial_fz;
	//			sum_virial_yz +=  py12 * Virial_fz;
	//		}

	//		// save virial
	//		virial_xx[tid1] += sum_virial_xx;
	//		virial_yy[tid1] += sum_virial_yy;
	//		virial_zz[tid1] += sum_virial_zz;
	//		virial_xy[tid1] += sum_virial_xy;
	//		virial_xz[tid1] += sum_virial_xz;
	//		virial_yz[tid1] += sum_virial_yz;

	//	}
	//}

	//__device__ rbmd::Real LJVirial(
	//		const rbmd::Real cut_off,
	//		const rbmd::Real px12,
	//		const rbmd::Real py12,
	//		const rbmd::Real pz12,
	//		const rbmd::Real eps_ij,
	//		const rbmd::Real sigma_ij)
	//{
	//	rbmd::Real virial_f = 0;;

	//	const rbmd::Real  small_value = 0.0001;
	//	const rbmd::Real dis_2 = px12 * px12 + py12 * py12 + pz12 * pz12;
	//	const rbmd::Real cut_off_2 = cut_off * cut_off;

	//	if (dis_2 < cut_off_2 && dis_2 > small_value)
	//	{
	//		rbmd::Real sigmaij_6 = sigma_ij * sigma_ij * sigma_ij * sigma_ij * sigma_ij * sigma_ij;
	//		rbmd::Real dis_6 = dis_2 * dis_2 * dis_2;
	//		rbmd::Real sigmaij_dis_6 = sigmaij_6 / dis_6;

	//		 virial_f = 0.5 * 24 * eps_ij * ((2 * sigmaij_dis_6 - 1) * sigmaij_dis_6) / dis_2;

	//}
	//	return virial_f;
	//}


	//
	void LJForceOp<device::DEVICE_GPU>::operator()(
						Box* box,
			            const rbmd::Real cut_off,
			            const rbmd::Id num_atoms,
		                const rbmd::Id* atoms_type,
			            const rbmd::Id* molecular_type,
			            const rbmd::Real* sigma,
			            const rbmd::Real* eps,
			            const rbmd::Id* start_id,
		                const rbmd::Id* end_id,
		                const rbmd::Id* id_verletlist,
			            const rbmd::Real* px,
			            const rbmd::Real* py,
			            const rbmd::Real* pz,
			            rbmd::Real* force_x,
			            rbmd::Real* force_y,
			            rbmd::Real* force_z,
			            rbmd::Real* evdwl,
						rbmd::Real* total_evdwl)
		{
		    unsigned int blocks_per_grid = (num_atoms + BLOCK_SIZE - 1) / BLOCK_SIZE;

		    CHECK_KERNEL(ComputeLJForce <<<blocks_per_grid, BLOCK_SIZE, 0, 0 >>> (box, cut_off, num_atoms, atoms_type, molecular_type,
				sigma, eps, start_id, end_id, id_verletlist, px, py, pz, force_x, force_y, force_z, evdwl,total_evdwl));
		}

	//RBL
	void LJRBLForceOp<device::DEVICE_GPU>::operator()(
		Box* box,
		const rbmd::Real rs,
		const rbmd::Real rc,
		const rbmd::Id pice_num,
		const rbmd::Id num_atoms,
		const rbmd::Id* atoms_type,
		const rbmd::Id* molecular_type,
		const rbmd::Real* sigma,
		const rbmd::Real* eps,
		const rbmd::Id* start_id,
		const rbmd::Id* end_id,
		const rbmd::Id* id_verletlist,
		const rbmd::Id* id_verletlist_rcs,
		const rbmd::Real* px,
		const rbmd::Real* py,
		const rbmd::Real* pz,
		rbmd::Real* corr_force_x,
		rbmd::Real* corr_force_y,
		rbmd::Real* corr_force_z,
		rbmd::Real* corr_value_x,
		rbmd::Real* corr_value_y,
		rbmd::Real* corr_value_z )
	{
		unsigned int blocks_per_grid = (num_atoms + BLOCK_SIZE - 1) / BLOCK_SIZE;

		CHECK_KERNEL(ComputeLJRBLForce <<<blocks_per_grid, BLOCK_SIZE, 0, 0 >>> (box, rs, rc, pice_num, num_atoms, atoms_type, molecular_type,
			sigma, eps, start_id, end_id, id_verletlist, id_verletlist_rcs, px, py, pz, corr_force_x, corr_force_y, corr_force_z, corr_value_x, corr_value_y, corr_value_z));
	}

	void ComputeChargeStructureFactorComponentOp<device::DEVICE_GPU>::operator()(
		const rbmd::Id num_atoms,
		const float3& k,
		const rbmd::Real* px,
		const rbmd::Real* py,
		const rbmd::Real* pz,
		const rbmd::Real* charge,
		rbmd::Real* density_real,
		rbmd::Real* density_imag)
	{
		unsigned int blocks_per_grid = (num_atoms + BLOCK_SIZE - 1) / BLOCK_SIZE;
		
		CHECK_KERNEL(ComputeChargeStructureFactorComponent <<<blocks_per_grid, BLOCK_SIZE, 0, 0 >>>
			(num_atoms, k, px, py ,pz, charge, density_real, density_imag));

	}

	void ComputeEwaldForceOp<device::DEVICE_GPU>::operator()(
		Box* box,
		const rbmd::Id num_atoms,
		const rbmd::Id  Kmax,
		const rbmd::Real alpha,
		const float2  whole_rhok,
		const rbmd::Real* charge,
		const rbmd::Real* px,
		const rbmd::Real* py,
		const rbmd::Real* pz,
		rbmd::Real* ewald_force_x,
		rbmd::Real* ewald_force_y,
		rbmd::Real* ewald_force_z)
	{
		unsigned int blocks_per_grid = (num_atoms + BLOCK_SIZE - 1) / BLOCK_SIZE;

		CHECK_KERNEL(ComputeEwaldForce <<<blocks_per_grid, BLOCK_SIZE, 0, 0 >>>
			(box,num_atoms, Kmax, alpha, whole_rhok, charge,
		     px, py, pz, ewald_force_x, ewald_force_y, ewald_force_z));

	}

}

